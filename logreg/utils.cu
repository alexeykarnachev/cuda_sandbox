#include "utils.h"

void* Managed::operator new(size_t size) {
    void* ptr;
    hipMallocManaged(&ptr, size);
    hipDeviceSynchronize();
    return ptr;
};

void Managed::operator delete(void* ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
}

float* create_random_data(size_t n_vals, int mod_div) {
    float* data = new float[n_vals];
    for (size_t i_val = 0; i_val < n_vals; ++i_val) {
        data[i_val] = (float)(rand() % mod_div);
    }
    return data;
}
