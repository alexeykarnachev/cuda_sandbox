#include "matrix.h"
#include <stdio.h>

Matrix::Matrix(size_t n_rows, size_t n_cols) : n_rows(n_rows), n_cols(n_cols) {
    size_t size = sizeof(float) * n_rows * n_cols;

    hipMallocManaged(&data, size);
    hipDeviceSynchronize();
};

void Matrix::set_data(float* source, size_t n_rows) {
    hipMemcpy(data, source, sizeof(float) * n_rows * n_cols, hipMemcpyHostToDevice);
}

Matrix::~Matrix() { hipFree(data); };

__host__ __device__ float& Matrix::operator()(size_t i_row, size_t i_col) {
    size_t i_val = i_row * this->n_cols + i_col;
    return this->data[i_val];
}
