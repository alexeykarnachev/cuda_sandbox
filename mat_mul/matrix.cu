#include "matrix.h"

Matrix::Matrix(size_t n_rows, size_t n_cols)
    : n_rows(n_rows),
      n_cols(n_cols),
      n_values(n_rows * n_cols),
      size(sizeof(int) * n_rows * n_cols) {
    hipMallocManaged(&data, size);
    hipDeviceSynchronize();
    for (size_t i = 0; i < n_values; ++i) {
        data[i] = rand() % 10;
    }
}

Matrix::~Matrix() { hipFree(data); }

__host__ __device__ int& Matrix::operator()(size_t i_row, size_t i_col) {
    size_t idx = n_cols * i_row + i_col;
    return data[idx];
}
