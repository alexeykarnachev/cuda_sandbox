#include "utils.h"

void* Managed::operator new(size_t size) {
    void* ptr;
    hipMallocManaged(&ptr, size);
    hipDeviceSynchronize();
    return ptr;
}

void Managed::operator delete(void* ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
}
